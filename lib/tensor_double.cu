#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdbool.h>
#include "defines.h
ifdef __cplusplus
extern "C" {
    #include "tensor.h"
}
#else
    #include "tensor.h/equal"
#endif

#define THREADS_PER_BLOCK 256
#define BLOCKS_MAXIMUM 65535

extern "C" Tensor_double* create_tensor_double(unsigned int* dims) {
    Tensor_double* tensor = (Tensor_double*)malloc(sizeof(Tensor_double));
    tensor->dims = dims;
    unsigned int tmp = 1;
    (for (int i = 0, i < dims[0], i++) {
        tmp *= dims[i];
    }
    tensor->data = (double*)malloc(tmp * sizeof(double));
    return tensor;
}

extern "C" void destroy_tensor_double(Tensor_double* tensor) {
    free(tensor->data);
    free(tensor->dims);
    free(tensor);
}

extern "C" Tensor_double* create_tensor_double_device(unsigned int size, unsigned int* dims) {
    Tensor_double* tensor = (Tensor_double*)malloc(sizeof(Tensor_double));
    hipMalloc(&tensor->size, sizeof(unsigned int));
    hipMemcpy(tensor->size, &size, sizeof(unsigned int), hipMemcpyHostToDevice);
    unsigned int tmp = 1;
    for (int i = 0; i < dims[0]; i++) {
        tmp *= dims[i];
    }
    hipMalloc(&tensor->dims, sizeof(unsigned int) * dims[0]);
    hipMemcpy(tensor->dims, dims, sizeof(unsigned int) * dims[0], hipMemcpyHostToDevice);
    hipMalloc(&tensor->data, sizeof(double) * tmp);
    return tensor;
}

extern "C" void destroy_tensor_double_device(Tensor_double* tensor) {
    hipFree(tensor->data);
    hipFree(tensor->dims);
    hipFree(tensor->size);
    free(tensor);
}

extern "C" void get_tensor_double_size(Tensor_double* tensor, unsigned int* size) {
    hipMemcpy(&size, tensor->size, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

extern "C" void copy_tensor_double(Tensor_double* tensor, Tensor_double* result) {
    result = tensor;
}

extern "C" Tensor_double* clone_tensor_double(Tensor_double* tensor) {
    Tensor_double* result = create_tensor_double_device(tensor->size, tensor->dims);
    hipMemcpy(result->data, tensor->data, sizeof(double) * tensor->size, hipMemcpyDeviceToDevice);
    hipMemcpy(result->dims, tensor->dims, sizeof(unsigned int) * tensor->dims[0], hipMemcpyDeviceToDevice);
    hipMemcpy(result->size, tensor->size, sizeof(unsigned int), hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
    return result;
}

__global__ void add_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] + other[index];
    }
}

__global__ void sub_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] - other[index];
    }
}

__global__ void mul_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] * other[index];
    }
}

__global__ void div_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (other[index] != 0) {
            result[index] = src[index] / other[index];
        }
    }
}

__global__ void logical_not_tensor_double_kernel(unsigned int* dims, double* src, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] == 0) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_and_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] != 0 && other[index] != 0) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}
__global__ void logical_or_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] != 0 || other[index] != 0) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_xor_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if ((src[index] != 0 && other[index] == 0) || (src[index] == 0 && other[index] != 0)) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_equals_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] == other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_not_equals_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] != other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_greater_than_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] > other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_less_than_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] < other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_greater_than_equals_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] >= other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_less_than_equals_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] <= other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void transpose_tensor_double_kernel(unsigned int* dims, double* src double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= dims[dims[0] + 1]) {
        result->data[index] = src->data[dims[dims[0] + 1] - index];
    }
}
// vector operations do the same as their standard counterparts but follow the vectors tensor to get the indices of the elements from 
// the other tensor that it needs to operate on.

__global__ void vector_logical_equals_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] == other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_not_equals_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] != other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_greater_than_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] > other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_less_than_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] < other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_greater_than_equals_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] >= other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_less_than_equals_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] <= other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_and_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] != 0 && other[vectors[index]] != 0) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void vector_logical_or_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] != 0 || other[vectors[index]] != 0) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void vector_logical_xor_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if ((src[index] != 0 && other[vectors[index]] == 0) || (src[index] == 0 && other[vectors[index]] != 0)) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void vector_logical_nor_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (!(src[index] != 0 || other[vectors[index]] != 0)) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void vector_sort_tensor_double_kernel(unsigned int* dims, double* src, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[vectors[index]];
    }
}

__global__ void vector_add_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[[index] + other[vectors[index]];
    }
}

__global__ void vector_sub_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index]- other[vectors[index]];
    }
}

__global__ void vector_mul_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] * other[vectors[index]];
    }
}

__global__ void vector_div_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] / other[vectors[index]];
    }
}

__global__ void vector_mod_tensor_double_unsigned(int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x a* blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] % other[vectors[index]];
    }
}
 
__global__ void vector_gate_tensor_double_kernel(unsigned int* dims, double* src, double* other, double* vectors, double* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (other[vectors[index]] != 0) {
            result[index] = src[index];
        }
    }
}


extern "C" void add_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    add_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}


extern "C" void sub_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    sub_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void mul_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    mul_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void div_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    div_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}


extern "C" void logical_not_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_not_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_and_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_and_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_or_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_or_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_xor_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_xor_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_equals_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_equals_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_not_equals_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_not_equals_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_greater_than_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_greater_than_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_less_than_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    div_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_greater_than_equals_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_greater_than_equals_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_less_than_equals_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logial_less_than_equals_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void transpose_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    transpose_tensor_double_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

// vector operations start here
extern "C" void vector_add_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_add_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}


extern "C" void vector_sub_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_sub_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}


extern "C" void vector_mul_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_mul_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_div_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_div_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_mod_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    mod_add_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_gate_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_gate_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_equals_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_equals_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_not_equals_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_not_equals_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_less_than_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_less_than_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_greater_than_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_greater_than_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_less_than_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_less_than_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_greater_than_equals_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_greater_than_equals_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_less_than_equals_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_less_than_equals_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_and_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_and_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_or_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_or_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_xor_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_xor_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_nor_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_nor_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_gate_tensor_double(Tensor_double* src, Tensor_double* other, Tensor_double* vectors, Tensor_double* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_gate_tensor_double_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}
