#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdbool.h>
#include "defines.h
ifdef __cplusplus
extern "C" {
    #include "tensor.h"
}
#else
    #include "tensor.h/equal"
#endif

#define THREADS_PER_BLOCK 256
#define BLOCKS_MAXIMUM 65535

extern "C" Tensor_short* create_tensor_short(unsigned int* dims) {
    Tensor_short* tensor = (Tensor_short*)malloc(sizeof(Tensor_short));
    tensor->dims = dims;
    unsigned int tmp = 1;
    (for (int i = 0, i < dims[0], i++) {
        tmp *= dims[i];
    }
    tensor->data = (short*)malloc(tmp * sizeof(short));
    return tensor;
}

extern "C" void destroy_tensor_short(Tensor_short* tensor) {
    free(tensor->data);
    free(tensor->dims);
    free(tensor);
}

extern "C" Tensor_short* create_tensor_short_device(unsigned int size, unsigned int* dims) {
    Tensor_short* tensor = (Tensor_short*)malloc(sizeof(Tensor_short));
    hipMalloc(&tensor->size, sizeof(unsigned int));
    hipMemcpy(tensor->size, &size, sizeof(unsigned int), hipMemcpyHostToDevice);
    unsigned int tmp = 1;
    for (int i = 0; i < dims[0]; i++) {
        tmp *= dims[i];
    }
    hipMalloc(&tensor->dims, sizeof(unsigned int) * dims[0]);
    hipMemcpy(tensor->dims, dims, sizeof(unsigned int) * dims[0], hipMemcpyHostToDevice);
    hipMalloc(&tensor->data, sizeof(short) * tmp);
    return tensor;
}

extern "C" void destroy_tensor_short_device(Tensor_short* tensor) {
    hipFree(tensor->data);
    hipFree(tensor->dims);
    hipFree(tensor->size);
    free(tensor);
}

extern "C" void get_tensor_short_size(Tensor_short* tensor, unsigned int* size) {
    hipMemcpy(&size, tensor->size, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

extern "C" void copy_tensor_short(Tensor_short* tensor, Tensor_short* result) {
    result = tensor;
}

extern "C" Tensor_short* clone_tensor_short(Tensor_short* tensor) {
    Tensor_short* result = create_tensor_short_device(tensor->size, tensor->dims);
    hipMemcpy(result->data, tensor->data, sizeof(short) * tensor->size, hipMemcpyDeviceToDevice);
    hipMemcpy(result->dims, tensor->dims, sizeof(unsigned int) * tensor->dims[0], hipMemcpyDeviceToDevice);
    hipMemcpy(result->size, tensor->size, sizeof(unsigned int), hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
    return result;
}

__global__ void add_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] + other[index];
    }
}

__global__ void sub_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] - other[index];
    }
}

__global__ void mul_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] * other[index];
    }
}

__global__ void div_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (other[index] != 0) {
            result[index] = src[index] / other[index];
        }
    }
}

__global__ void logical_not_tensor_short_kernel(unsigned int* dims, short* src, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] == 0) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_and_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] != 0 && other[index] != 0) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}
__global__ void logical_or_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] != 0 || other[index] != 0) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_xor_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if ((src[index] != 0 && other[index] == 0) || (src[index] == 0 && other[index] != 0)) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_equals_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] == other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_not_equals_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] != other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_greater_than_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] > other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_less_than_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] < other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_greater_than_equals_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] >= other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void logical_less_than_equals_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] <= other[index]) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void transpose_tensor_short_kernel(unsigned int* dims, short* src short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= dims[dims[0] + 1]) {
        result->data[index] = src->data[dims[dims[0] + 1] - index];
    }
}
// vector operations do the same as their standard counterparts but follow the vectors tensor to get the indices of the elements from 
// the other tensor that it needs to operate on.

__global__ void vector_logical_equals_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] == other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_not_equals_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] != other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_greater_than_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] > other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_less_than_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] < other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_greater_than_equals_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] >= other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_less_than_equals_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = (src[index] <= other[vectors[index]]) ? 1 : 0;
    }
}

__global__ void vector_logical_and_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] != 0 && other[vectors[index]] != 0) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void vector_logical_or_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (src[index] != 0 || other[vectors[index]] != 0) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void vector_logical_xor_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if ((src[index] != 0 && other[vectors[index]] == 0) || (src[index] == 0 && other[vectors[index]] != 0)) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void vector_logical_nor_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (!(src[index] != 0 || other[vectors[index]] != 0)) {
            result[index] = 1;
        } else {
            result[index] = 0;
        }
    }
}

__global__ void vector_sort_tensor_short_kernel(unsigned int* dims, short* src, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[vectors[index]];
    }
}

__global__ void vector_add_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[[index] + other[vectors[index]];
    }
}

__global__ void vector_sub_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index]- other[vectors[index]];
    }
}

__global__ void vector_mul_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] * other[vectors[index]];
    }
}

__global__ void vector_div_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] / other[vectors[index]];
    }
}

__global__ void vector_mod_tensor_short_unsigned(int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x a* blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        result[index] = src[index] % other[vectors[index]];
    }
}
 
__global__ void vector_gate_tensor_short_kernel(unsigned int* dims, short* src, short* other, short* vectors, short* result) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dims[dims[0] + 1]) {
        if (other[vectors[index]] != 0) {
            result[index] = src[index];
        }
    }
}


extern "C" void add_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    add_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}


extern "C" void sub_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    sub_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void mul_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    mul_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void div_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    div_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}


extern "C" void logical_not_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_not_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_and_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_and_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_or_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_or_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_xor_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_xor_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_equals_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_equals_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_not_equals_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_not_equals_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_greater_than_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_greater_than_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_less_than_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    div_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_greater_than_equals_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logical_greater_than_equals_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void logical_less_than_equals_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    logial_less_than_equals_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void transpose_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    cudamemcpy(dims, src->dims[0], sizeof(unsigned int) * (2 + dim0), cudaMemcpymdeviceToHost);
    transpose_tensor_short_kernel<<<(dims[dims[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, result->data);
    hipDeviceSynchronize();
}

// vector operations start here
extern "C" void vector_add_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_add_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}


extern "C" void vector_sub_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_sub_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}


extern "C" void vector_mul_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_mul_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_div_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_div_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_mod_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    mod_add_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_gate_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_gate_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_equals_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_equals_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_not_equals_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_not_equals_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_less_than_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_less_than_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_greater_than_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_greater_than_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_less_than_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_less_than_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_greater_than_equals_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_greater_than_equals_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_less_than_equals_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_less_than_equals_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_and_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_and_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_or_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_or_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_xor_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_xor_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_logical_nor_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_logical_nor_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}

extern "C" void vector_gate_tensor_short(Tensor_short* src, Tensor_short* other, Tensor_short* vectors, Tensor_short* result) {
    unsigned int dim0;
    hipMemcpy(dim0, src->dims[0], sizeof(unsigned int), cudamemcpyDeviceToHost);
    unsigned int* dims = (unsigned int*) malloc(sizeof(unsigned int) * (2 + dim0));
    hipMemcpy(dims, src->dims[0],sizeof(unsigned int) * (2 + dim0) hipMemcpyDeviceToHost);
    vector_gate_tensor_short_kernel<<<(dims[dime[0] + 1] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(short)>>>(src->dims, src->data, other->data, vectors->data, result->data);
    hipDeviceSynchronize();
}
