#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include "defines.h"
#ifdef __cplusplus
extern "C" {
    #include "tensor.h"
}
#else
    #include "tensor.h"
#endif
#ifdef __DEBUG__
#endif

Tensor* create_tensor(int size) {
    Tensor *tensor = (Tensor*) malloc(sizeof(Tensor));
    tensor->size = (int*) malloc(sizeof(int));
    tensor->size = &size;
    tensor->data = (short*) malloc(size*sizeof(short)); 
    return tensor;
}

void destroy_tensor(Tensor* tensor) {
    free(tensor->data);
    free(tensor->size);
    free(tensor);
    tensor = NULL;
}

void set_cpu_to_device_tensor(Tensor* tensor) {
    Tensor* d_tensor;
    hipMalloc((void **) &d_tensor->size, sizeof(int));
    hipMemcpy(&d_tensor->size, &tensor->size, sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **) &d_tensor->data, tensor->size[0] * sizeof(short));
    hipMemcpy(d_tensor->data, tensor->data, tensor->size[0] * sizeof(short), hipMemcpyHostToDevice);
    free(tensor->data);
    free(tensor->size);
    tensor = d_tensor;
    d_tensor = NULL;
}

void set_device_to_cpu_tensor(Tensor* tensor) {
    Tensor* c_tensor = (Tensor*) malloc(sizeof(Tensor*));
    c_tensor->size = (int*) malloc(sizeof(int));
    hipMemcpy(&c_tensor->size, &tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    c_tensor->data = (short*) malloc(sizeof(short) * c_tensor->size[0]);
    hipMemcpy(&c_tensor->data, &tensor->data, sizeof(short) * c_tensor->size[0], hipMemcpyDeviceToHost);
    hipFree(tensor->size);
    hipFree(tensor->data);
    tensor = c_tensor;
    c_tensor = NULL;
}

void copy_device_to_device_tensor(Tensor* tensor, Tensor* result) {
    hipMemcpy(&result->size, &tensor->size, sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(&result->data, &tensor->data, sizeof(short) * tensor->size[0], hipMemcpyDeviceToDevice);
}

void copy_device_to_cpu_tensor(Tensor* tensor, Tensor* result) {
    hipMemcpy(&result->size, &tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&result->data, &tensor->data, sizeof(short) * tensor->size[0], hipMemcpyDeviceToHost);
}

void copy_cpu_to_device_tensor(Tensor* tensor, Tensor* result) {
    hipMemcpy(&result->size, &tensor->size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&result->data, &tensor->data, sizeof(short) * tensor->size[0], hipMemcpyHostToDevice);
}

void copy_cpu_to_cpu_tensor(Tensor* tensor, Tensor* result) {
    memcpy(&result->size[0], &tensor->size[0], sizeof(int));
    memcpy(&result->data, &tensor->data, sizeof(short) * tensor->size[0]);
}

Tensor* create_device_tensor(int size) {
    Tensor *tensor = (Tensor*) malloc(sizeof(Tensor));
    hipMalloc((void **) &tensor->size, sizeof(int));
    hipMemcpy(&tensor->size, &size, sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **) &tensor->data, size * sizeof(short));
    return tensor;
}

void destroy_device_tensor(Tensor* tensor) {
    hipFree(&tensor->data);
    hipFree(&tensor->size);
    tensor = NULL;
}

int init_tensor(Tensor* tensor, int size) {
    tensor->size = &size;
    tensor->data = (short*)malloc(size * sizeof(short));
    if (!tensor->data) {
        return -1;
    }
    return 0;
}

__global__
void vector_sort_tensor(Tensor* tensor,  Tensor* vectors,  Tensor* result) {
    if (vectors->size[0] != tensor->size[0] || result->size[0] != tensor->size[0]) {
        printf("Error: index out of bounds\n");
        return; 
    }
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < tensor->size[0]; i+= stride) {
        result->data[i] = tensor->data[vectors->data[i]];
    }
}

__global__
void vector_add_tensor(Tensor* tensor, Tensor* other, Tensor* vectors, Tensor* result) {
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = 0; i < tensor->size[0]; i++) {
        result->data[i] = tensor->data[i] + other->data[vectors->data[i]];
    }
}

__global__
void vector_sub_tensor(Tensor* tensor, Tensor* other, Tensor* vectors, Tensor* result) {
    for (int i = 0; i < tensor->size[0]; i++) {
        result->data[i] = tensor->data[i] - other->data[vectors->data[i]];
    }
}

__global__
void vector_mul_tensor(Tensor* tensor, Tensor* other, Tensor* vectors, Tensor* result) {
    for (int i = 0; i < tensor->size[0]; i++) {
        result->data[i] = tensor->data[i] * other->data[vectors->data[i]];
    }
}

__global__
void vector_div_tensor(Tensor* tensor, Tensor* other, Tensor* vectors, Tensor* result) {
    for (int i = 0; i < tensor->size[0]; i++) {
        result->data[i] = tensor->data[i] / other->data[vectors->data[i]];
    }
}

__global__
void vector_gate_tensor(Tensor* tensor, Tensor* booleans, Tensor* vectors, Tensor* result) {
    for (int i = 0; i < tensor->size[0]; i++) {
        if (booleans->data[i] == 1) {
            result->data[i] = tensor->data[vectors->data[i]];
        }
    }
}

__global__
void set_tensor( Tensor* tensor, int index, short value) {
    if (index > tensor->size[0]) {
        printf("Error: index out of bounds\n");
        return;
    }
    tensor->data[index] = value;
}       

        __global__
        void zeros_tensor( Tensor* tensor) {

            int index = threadIdx.x;
            tensor->data[index] = 0;
        }

        __global__
        void ones_tensor( Tensor* tensor) {
            int index = threadIdx.x;
            tensor->data[index] = 1;
        }

        void rm_Tensor( Tensor* tensor) {
            hipFree(&tensor->data);
        }

        __global__
        void print_tensor( Tensor* tensor) {

            int index = threadIdx.x;
            int stride = blockDim.x;
            printf("Value at index: %d is %d\n", index, tensor->data[index]);
        }

        __global__
        void fill_tensor( Tensor* tensor, int value) {
            int index = threadIdx.x;
            tensor->data[index] = value;
        }

        __global__
        void add_tensor( Tensor* tensor,  Tensor* other,  Tensor* result) {
            int index = threadIdx.x;
            result->data[index] = tensor->data[index], other->data[index];
        }

        __global__
        void sub_tensor( Tensor* tensor,  Tensor* other,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            if (tensor->size[0] != other->size[0]) {
                printf("Error: size mismatch\n");
                return;
            }
            for (int i = index; i < tensor->size[0]; i+= stride) {
                result->data[i] = tensor->data[i] - other->data[i];
            }
        }

        __global__
        void mul_tensor( Tensor* tensor,  Tensor* other,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            if (tensor->size[0] != other->size[0]) {
                printf("Error: size mismatch\n");
                return;
            }
            for (int i = index; i < tensor->size[0]; i+= stride) {
                result->data[i] = tensor->data[i] * other->data[i];
            }
        }

        __global__
        void div_tensor( Tensor* tensor,  Tensor* other,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            if (tensor->size[0] != other->size[0]) {
                printf("Error: size mismatch\n");
                return;
            }
            for (int i = index; i < tensor->size[0]; i+= stride) {
                result->data[i] = tensor->data[i] / other->data[i];
            }
        }

        __global__
        void add_scalar_tensor( Tensor* tensor, int value,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            for (int i = index; i < tensor->size[0]; i+= stride) {
                result->data[i] = tensor->data[i] + value;
            }
        }

        __global__
        void sub_scalar_tensor( Tensor* tensor, int value,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            for (int i = index; i < tensor->size[0]; i+= stride) {
                result->data[i] = tensor->data[i] - value;
            }
        }

        __global__
        void mul_scalar_tensor( Tensor* tensor, int value,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            for (int i = index; i < tensor->size[0]; i+= stride) {
                result->data[i] = tensor->data[i] * value;
            }
        }

        __global__
        void div_scalar_tensor( Tensor* tensor, int value,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            for (int i = index; i < tensor->size[0]; i+= stride) {
                result->data[i] = tensor->data[i] / value;
            }
        }

        __global__
        void transpose_tensor( Tensor* tensor,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            for (int i = index; i < tensor->size[0]; i+= stride) {
                result->data[i] = tensor->data[tensor->size[0] - i - 1];
            }
        }

        __global__
        void sum_tensor( Tensor* tensor,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            for (int i = index; i < tensor->size[0]; i+= stride) {
                result->data[0] += tensor->data[i];
            }
        }

        __global__
        void mean_tensor( Tensor* tensor,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            for (int i = index; i < tensor->size[0]; i+= stride) {
                result->data[0] += tensor->data[i];
            }
            result->data[0] /= tensor->size[0];
        }

        __global__
        void max_tensor( Tensor* tensor,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            for (int i = index; i < tensor->size[0]; i+= stride) {
                if (tensor->data[i] > result->data[0]) {
                    result->data[0] = tensor->data[i];
                }
            }
        }

        __global__
        void min_tensor( Tensor* tensor,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            for (int i = index; i < tensor->size[0]; i+= stride) {
                if (tensor->data[i] < result->data[0]) {
                    result->data[0] = tensor->data[i];
                }
            }
        }

        __global__
        void gradient_tensor( Tensor* tensor,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            for (int i = index; i < tensor->size[0]; i+= stride) {
                result->data[i] = tensor->data[i + 1] - tensor->data[i];
            }
        }

        __global__
        void gate_tensor( Tensor* tensor,  Tensor* bools,  Tensor* result) {
            int index = threadIdx.x;
            int stride = blockDim.x;
            for (int i = index; i < tensor->size[0]; i+= stride) {
                if (bools->data[i] == 1) {
                    result->data[i] = tensor->data[i];
                }
            }
        }

         bool check_size( Tensor* tensor,  Tensor* other) {
            if (tensor->size[0] != other->size[0]) {
                printf("Error: size mismatch\n");
                return false;
            }
            if (sizeof(&tensor->data) != sizeof(&other->data)) {
                printf("Error: size mismatch\n");
                return false;
            }
            return true;
        }

        extern "C" bool check_size_3( Tensor* tensor,  Tensor* other,  Tensor* result) {
            if (tensor->size[0] != other->size[0] || tensor->size[0] != result->size[0]) {
                printf("Error: size mismatch\n");
                return false;
            }
            if (sizeof(&tensor->data) != sizeof(&other->data) || sizeof(&tensor->data) != sizeof(&result->data)) {
                printf("Error: size mismatch\n");
                return false;
            }
            return true;
        }

extern "C" void vector_add_wrapper( Tensor* tensor,  Tensor* other,  Tensor* vectors,  Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size < 256 == 0) {
        vector_add_tensor<<<1, 256>>>(tensor, other, vectors, result);
    } else if (size / 256 < 8) {
        vector_add_tensor<<<(size / 256), 256>>>(tensor, other, vectors, result);
    } else {
        vector_add_tensor<<<7, 256>>>(tensor, other, vectors, result);
    }
    hipDeviceSynchronize();
}

extern "C" void vector_sub_wrapper( Tensor* tensor,  Tensor* other,  Tensor* vectors,  Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size < 256 == 0) {
        vector_sub_tensor<<<1, 256>>>(tensor, other, vectors, result);
    } else if (size / 256 < 8) {
        vector_sub_tensor<<<(size / 256), 256>>>(tensor, other, vectors, result);
    } else {
        vector_sub_tensor<<<7, 256>>>(tensor, other, vectors, result);
    }
    hipDeviceSynchronize();
}

extern "C" void vector_mul_wrapper( Tensor* tensor,  Tensor* other,  Tensor* vectors,  Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size < 256 == 0) {
        vector_mul_tensor<<<1, 256>>>(tensor, other, vectors, result);
    } else if (size / 256 < 8) {
        vector_mul_tensor<<<(size / 256), 256>>>(tensor, other, vectors, result);
    } else {
        vector_mul_tensor<<<7, 256>>>(tensor, other, vectors, result);
    }
    hipDeviceSynchronize();
}

extern "C" void vector_div_wrapper( Tensor* tensor,  Tensor* other,  Tensor* vectors,  Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size < 256 == 0) {
        vector_div_tensor<<<1, 256>>>(tensor, other, vectors, result);
    } else if (size / 256 < 8) {
        vector_div_tensor<<<(size / 256), 256>>>(tensor, other, vectors, result);
    } else {
        vector_div_tensor<<<7, 256>>>(tensor, other, vectors, result);
    }
    hipDeviceSynchronize();
}

extern "C" void vector_gate_wrapper( Tensor* tensor,  Tensor* booleans,  Tensor* vectors,  Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size < 256 == 0) {
        vector_gate_tensor<<<1, 256>>>(tensor, booleans, vectors, result);
    } else if (size / 256 < 8) {
        vector_gate_tensor<<<(size / 256), 256>>>(tensor, booleans, vectors, result);
    } else {
        vector_gate_tensor<<<7, 256>>>(tensor, booleans, vectors, result);
    }
    hipDeviceSynchronize();
}

extern "C" void zeros_tensor_wrapper( Tensor* tensor) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    zeros_tensor<<<1,size>>>(tensor);
    hipDeviceSynchronize();
}

extern "C" void ones_tensor_wrapper( Tensor* tensor) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    ones_tensor<<<1, size>>>(tensor);
    hipDeviceSynchronize();
}

extern "C" void vector_sort_tensor_wrapper( Tensor* tensor,  Tensor* vectors,  Tensor* result)  {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);

   if (size / 256 == 0) {
        vector_sort_tensor<<<1, 256>>>(tensor, vectors, result);
    } else if (size / 256 < 8) {
        vector_sort_tensor<<<(size / 256), 256>>>(tensor, vectors, result);
    } else {
        vector_sort_tensor<<<7, 256>>>(tensor, vectors, result);
    }
    hipDeviceSynchronize();
}

extern "C" void print_tensor_wrapper( Tensor* tensor) {
     int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    print_tensor<<<1, size>>>(tensor);
    hipDeviceSynchronize();
}

extern "C" void fill_tensor_wrapper( Tensor* tensor, int value) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        fill_tensor<<<1, 256>>>(tensor, value);
    } else if (size / 256 < 8) {
        fill_tensor<<<(size / 256), 256>>>(tensor, value);
    } else {
        fill_tensor<<<7, 256>>>(tensor, value);
    }
    hipDeviceSynchronize();
    hipFree(tensor);
}

extern "C" void add_tensor_wrapper( Tensor* tensor,  Tensor* other,  Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    add_tensor<<<1, size>>>(tensor, other, result);
    hipDeviceSynchronize();
}

extern "C" void sub_tensor_wrapper( Tensor* tensor,  Tensor* vectors,  Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
          sub_tensor<<<1, 256>>>(tensor, vectors, result);
     } else if (size / 256 < 8) {
          sub_tensor<<<(size / 256), 256>>>(tensor, vectors, result);
     } else {
          sub_tensor<<<7, 256>>>(tensor, vectors, result);
     }
    hipDeviceSynchronize();
}

extern "C" void mul_tensor_wrapper( Tensor* tensor,  Tensor* vectors,  Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
          mul_tensor<<<1, 256>>>(tensor, vectors, result);
     } else if (size / 256 < 8) {
          mul_tensor<<<(size / 256), 256>>>(tensor, vectors, result);
     } else {
          mul_tensor<<<7, 256>>>(tensor, vectors, result);
     }
    hipDeviceSynchronize();
}

extern "C" void div_tensor_wrapper( Tensor* tensor,  Tensor* vectors,  Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
          div_tensor<<<1, 256>>>(tensor, vectors, result);
     } else if (size / 256 < 8) {
          div_tensor<<<(size / 256), 256>>>(tensor, vectors, result);
     } else {
          div_tensor<<<7, 256>>>(tensor, vectors, result);
     }
    hipDeviceSynchronize();
}

extern "C" void add_scalar_tensor_wrapper(Tensor* tensor, int value, Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        add_scalar_tensor<<<1, 256>>>(tensor, value, result);
    } else if (size / 256 < 8) {
        add_scalar_tensor<<<(size / 256), 256>>>(tensor, value, result);
    } else {
        add_scalar_tensor<<<7, 256>>>(tensor, value, result);
    }
    hipDeviceSynchronize();
}

extern "C" void sub_scalar_tensor_wrapper(Tensor* tensor, int value, Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        sub_scalar_tensor<<<1, 256>>>(tensor, value, result);
    } else if (size / 256 < 8) {
        sub_scalar_tensor<<<(size / 256), 256>>>(tensor, value, result);
    } else {
        sub_scalar_tensor<<<7, 256>>>(tensor, value, result);
    }
    hipDeviceSynchronize();
}

extern "C" void mul_scalar_tensor_wrapper(Tensor* tensor, int value, Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        mul_scalar_tensor<<<1, 256>>>(tensor, value, result);
    } else if (size / 256 < 8) {
        mul_scalar_tensor<<<(size / 256), 256>>>(tensor, value, result);
    } else {
        mul_scalar_tensor<<<7, 256>>>(tensor, value, result);
    }
    hipDeviceSynchronize();
}

extern "C" void div_scalar_tensor_wrapper(Tensor* tensor, int scalar, Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        div_scalar_tensor<<<1, 256>>>(tensor, scalar, result);
    } else if (size / 256 < 8) {
        div_scalar_tensor<<<(size / 256), 256>>>(tensor, scalar, result);
    } else {
        div_scalar_tensor<<<7, 256>>>(tensor, scalar, result);
    }
    hipDeviceSynchronize();
}

extern "C" void transpose_tensor_wrapper(Tensor* tensor, Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        transpose_tensor<<<1, 256>>>(tensor, result);
    } else if (size / 256 < 8) {
        transpose_tensor<<<(size / 256), 256>>>(tensor, result);
    } else {
        transpose_tensor<<<7, 256>>>(tensor, result);
    }
    hipDeviceSynchronize();
}

extern "C" void sum_tensor_wrapper(Tensor* tensor, Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        sum_tensor<<<1, 256>>>(tensor, result);
    } else if (size / 256 < 8) {
        sum_tensor<<<(size / 256), 256>>>(tensor, result);
    } else {
        sum_tensor<<<7, 256>>>(tensor, result);
    }
    hipDeviceSynchronize();
}

extern "C" void mean_tensor_wrapper(Tensor* tensor, Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        mean_tensor<<<1, 256>>>(tensor, result);
    } else if (size / 256 < 8) {
        mean_tensor<<<(size / 256), 256>>>(tensor, result);
    } else {
        mean_tensor<<<7, 256>>>(tensor, result);
    }
    hipDeviceSynchronize();
}

extern "C" void max_tensor_wrapper(Tensor* tensor, Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        max_tensor<<<1, 256>>>(tensor, result);
    } else if (size / 256 < 8) {
        max_tensor<<<(size / 256), 256>>>(tensor, result);
    } else {
        max_tensor<<<7, 256>>>(tensor, result);
    }
}

extern "C" void min_tensor_wrapper(Tensor* tensor, Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        min_tensor<<<1, 256>>>(tensor, result);
    } else if (size / 256 < 8) {
        min_tensor<<<(size / 256), 256>>>(tensor, result);
    } else {
        min_tensor<<<7, 256>>>(tensor, result);
    }
}

extern "C" void gradient_tensor_wrapper(Tensor* tensor, Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        gradient_tensor<<<1, 256>>>(tensor, result);
    } else if (size / 256 < 8) {
        gradient_tensor<<<(size / 256), 256>>>(tensor, result);
    } else {
        gradient_tensor<<<7, 256>>>(tensor, result);
    }
}

extern "C" void gate_tensor_wrapper(Tensor* tensor, Tensor* booleans, Tensor* result) {
    int size;
    hipMemcpy(&size, tensor->size, sizeof(int), hipMemcpyDeviceToHost);
    if (size / 256 == 0) {
        gate_tensor<<<1, 256>>>(tensor, booleans, result);
    } else if (size / 256 < 8) {
        gate_tensor<<<(size / 256), 256>>>(tensor, booleans, result);
    } else {
        gate_tensor<<<7, 256>>>(tensor, booleans, result);
    }
}
